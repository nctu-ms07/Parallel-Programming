#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void mandelKernel(const float x0, const float y0,
                             const float dx, const float dy,
                             const int width,
                             const int count,
                             int *const d_img,
                             const size_t pitch) {
  // To avoid error caused by the floating number, use the following pseudo code
  //
  // float x = lowerX + thisX * stepX;
  // float y = lowerY + thisY * stepY;

  int loop_i = blockIdx.x * blockDim.x + threadIdx.x;
  int loop_j = blockIdx.y * blockDim.y + threadIdx.y;

  float c_re = x0 + loop_i * dx;
  float c_im = y0 + loop_j * dy;

  float z_re = c_re, z_im = c_im;
  int i;
  float new_re, new_im;
  for (i = 0; i < count; ++i) {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    new_re = z_re * z_re - z_im * z_im;
    new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  // int index = (loop_j * width + loop_i);
  *((int *) ((char *) d_img + loop_j * pitch) + loop_i) = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;

  const int size = resX * resY * sizeof(int);
  size_t pitch;
  int *h_img, *d_img;
  hipHostAlloc(&h_img, size, hipHostMallocDefault);
  hipMallocPitch(&d_img, &pitch, resX * sizeof(int), resY);

  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(resX / block.x, resY / block.y);
  mandelKernel<<<grid, block>>>(lowerX, lowerY, stepX, stepY, resX, maxIterations, d_img, pitch);

  hipMemcpy2D(h_img, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, hipMemcpyDefault);
  hipFree(d_img);
  memcpy(img, h_img, size);
  hipHostFree(h_img);
}
